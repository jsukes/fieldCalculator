
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>

// consider making wfs into a float4 array, with the wfs from 4 els stacked side-by-side -> less loop iters

// command to check gpu device info
/*** ./usr/local/cuda-9.2/extras/demo_suite/deviceQuery ***/

// compile command
/*** nvcc -Xptxas=-O3,-v -use_fast_math -maxrregcount=20 fieldMaker_paramTester.cu ***/

// physical constants
#define CW				1.482	// water sound speed (mm/us)

// receive system specifics
#define RECLEN			8191	// length of acquired signal
#define NELS			160		// number of receiver elements
#define NLOCS			5
#define NPULSES			4
#define ADC_CLOCK		20		// digitizer clock (MHz)

// prescribed constant
#define DT				105.0	// time-of-flight = ( element-to-origin (100 us) + transmit-system-specific delays (10.5 us))

// derived constants to take some load off gpu
#define DT_TIMES_ADC_CLOCK	( DT * ADC_CLOCK )
#define ADC_CLOCK_OVER_CW	( ADC_CLOCK / CW )

//~ const float ADC_CLOCK_OVER_CW = ADC_CLOCK/CW;
//~ const float DT_TIMES_ADC_CLOCK = DT*ADC_CLOCK;

// center point of the pressure field being calculated
#define FIELD_X0 		0.0
#define FIELD_Y0 		0.0
#define FIELD_Z0 		0.0

// size of the pressure field to calculate ( units = mm ) 
#define FIELD_DIM_X 	30.0
#define FIELD_DIM_Y 	30.0
#define FIELD_DIM_Z 	30.0

// constants for gpu
#define BLOCK_DIM_X		4
#define BLOCK_DIM_Y		8
#define BLOCK_DIM_Z		8

// max threads = 1024 (GPU specific, this is for laptop) [8*8*16=1024]
#define THREAD_DIM_X 	16
#define THREAD_DIM_Y 	8
#define THREAD_DIM_Z	8 //works best if THREAD_DIM_Z < THREAD_DIM_X/Y
#define BLOCK_SIZE		(THREAD_DIM_X*THREAD_DIM_Y*THREAD_DIM_Z)

// constant to calculate indices of calculation grid
#define IE (BLOCK_DIM_X*THREAD_DIM_X)
#define JE (BLOCK_DIM_Y*THREAD_DIM_Y)
#define KE (BLOCK_DIM_Z*THREAD_DIM_Z)

// variable to hold coordinate locations of array elements
__constant__ float4 arxyz[NELS];

// projects the measured signals from the array elements back into the field, sums them together
__global__ void getSignalIntensityField(float *si_field, float4 *wfs, float4 *XYZ, float dt){
	
	int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
	int z = __mul24(blockIdx.z,blockDim.z) + threadIdx.z;	
	int sigint_idx = (z*JE + y)*IE + x; // index of field location being calculated
		
	int tidx; 						// index of 'wfs' vector
	float dx,dy,dz;					// unit distances from array element to location in calculation field 
	float4 xyz = XYZ[sigint_idx];	// makes calculation faster to load value stored in global mem into local mem 

	#pragma unroll 20
	for(int eln = 0; eln<NELS; eln++){
		dx =  __fsub_rn(xyz.x,arxyz[eln].x);
		dy =  __fsub_rn(xyz.y,arxyz[eln].y);
		dz =  __fsub_rn(xyz.z,arxyz[eln].z);
				
		tidx = __float2int_rn( __fadd_rn( __fmul_rn( __fsqrt_rn( __fadd_rn( __fadd_rn( __fmul_rn(dx,dx) , __fmul_rn(dy,dy) ) , __fmul_rn(dz,dz)) ) , ADC_CLOCK_OVER_CW), (DT+dt)*ADC_CLOCK)) + __mul24(eln,RECLEN);


		// this script loads 4 waveforms at a time, this adds the signal values from all of them together at once
		si_field[sigint_idx] += __fadd_rn(__fsub_rn(wfs[ tidx ].x, wfs[ tidx ].y),__fsub_rn(wfs[ tidx ].z, wfs[ tidx ].w));	
	}	
}


// function to set the field to 0
__global__ void resetSignalIntensityField(float *pfield){
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;	
	int sigint_idx = z*IE*JE + y*IE + x;	
	pfield[sigint_idx] = 0;
}

// loads the signals from file
void loadWF(float4 *wf){
	float *wftmp;
	wftmp = (float *)malloc(RECLEN*NELS*NPULSES*sizeof(float));
	FILE *fname;			
	//~ fname = fopen("hiampwfHilb_sk","rb");
	fname = fopen("hiampwf_sk","rb");
	fread(wftmp,sizeof(wftmp),RECLEN*NELS*NPULSES,fname);
	fclose(fname);
	
	for(int rl=1;rl<(NPULSES*NELS*RECLEN);rl++){
		wf[rl].x=wftmp[rl];
		wf[rl].y=wftmp[rl+NELS*RECLEN]*0;
		wf[rl].z=wftmp[rl+(NELS*RECLEN)*2]*0;
		wf[rl].w=wftmp[rl+3*NELS*RECLEN]*0;
	}
	
	for(int n=0;n<NELS;n++){
		wf[n*RECLEN].x=0;
		wf[n*RECLEN].y=0;
		wf[n*RECLEN].z=0;
		wf[n*RECLEN].w=0;		
	}
	free(wftmp);
}

// loads the coordinates of the receiver elements of the array
void loadArray(){	
	float *array_tmp;
	array_tmp = (float *)malloc(256*3*sizeof(float));
	float4 arxyz_h[NELS];
	FILE *fname;				
	fname = fopen("./dataFiles/arrayCoords_bin","rb");	
	fread(array_tmp,sizeof(array_tmp),RECLEN*NELS,fname);
	fclose(fname);	
	
	int cntr1, cntr2;
	cntr1 = 0;
	for(cntr2=0;cntr2<256;cntr2++){		
		if( (cntr2<128)  && (cntr2%4 == 0) ){
			arxyz_h[cntr1].x = array_tmp[cntr2];
			arxyz_h[cntr1].y = array_tmp[cntr2+256];
			arxyz_h[cntr1].z = array_tmp[cntr2+512];
			cntr1++;
		} else if (cntr2>=128) {
			arxyz_h[cntr1].x = array_tmp[cntr2];
			arxyz_h[cntr1].y = array_tmp[cntr2+256];
			arxyz_h[cntr1].z = array_tmp[cntr2+512];
			cntr1++;
		}	
	}
	hipMemcpyToSymbol(HIP_SYMBOL(arxyz),arxyz_h,NELS*sizeof(float4));
	free(array_tmp);
}

// generates a 4-vector storing the locations of all points in the field we are trying to calculate
// the 4 field of the vector is unused, but in my testing it looked like it was faster to use float4 than float3 and I wasn't running up against any memory limits so I stuck with it
void setFieldLocsXYZ(float4 *xyz){
	int cntr1,cntr2,cntr3;
	int xsteps,ysteps,zsteps;

	xsteps = (BLOCK_DIM_X*THREAD_DIM_X);
	ysteps = (BLOCK_DIM_Y*THREAD_DIM_Y);
	zsteps = (BLOCK_DIM_Z*THREAD_DIM_Z);
	
	float x0,y0,z0;
	x0 = FIELD_X0 - FIELD_DIM_X/2.0;
	y0 = FIELD_Y0 - FIELD_DIM_Y/2.0;
	z0 = FIELD_Z0 - FIELD_DIM_Z/2.0;
	
	float dx,dy,dz;
	dx = FIELD_DIM_X/xsteps;
	dy = FIELD_DIM_Y/ysteps;
	dz = FIELD_DIM_Z/zsteps;
	
	for(cntr1=0;cntr1<xsteps;cntr1++){	
		for(cntr2=0;cntr2<ysteps;cntr2++){
			for(cntr3=0;cntr3<ysteps;cntr3++){
				xyz[(cntr1*ysteps+cntr2)*zsteps+cntr3].x = x0+cntr3*dx;
				xyz[(cntr1*ysteps+cntr2)*zsteps+cntr3].y = y0+cntr2*dy;
				xyz[(cntr1*ysteps+cntr2)*zsteps+cntr3].z = z0+cntr1*dz;
			}
		}
	}	
}


int plotData(float *siField, FILE *pipe, float dt){
	//~ printf("HELLLO\n");
	FILE *fname;			
	fname = fopen("fdata.dat","w");
	
	int nz = 3;
	int N = 5;
	int xx,yy,zz;
	xx=0; yy=0;
	
	for(int nn=0;nn<nz;nn++){
		for(yy=0;yy<BLOCK_DIM_Y*THREAD_DIM_Y;yy++){
			for(zz=0;zz<N;zz++){
				for(xx=0;xx<BLOCK_DIM_X*THREAD_DIM_X;xx++){	
					fprintf(fname,"%f ",siField[(((zz+nn*N)*BLOCK_DIM_Z*THREAD_DIM_Z/(N*nz))*JE + yy)*IE + xx]);
				}			
			}
			fprintf(fname,"\n");	
		}
	}
	
	fclose(fname);
	usleep(35000);
	fprintf(pipe, "set term x11 size 3000,1800 font 'Helvetica,80'\n");
	fprintf(pipe, "set lmargin at screen 0.02; set rmargin at screen 0.9; set bmargin at screen 0.02; set tmargin at screen 0.98\n");
	fprintf(pipe, "set pm3d map\n");
	fprintf(pipe, "set xrange [%.2f:%.2f]; set yrange [%.2f:%.2f];\n", (0.0-FIELD_DIM_X*N*0.5),(FIELD_DIM_X*N*0.5),(0.0-FIELD_DIM_Y*nz*0.5),(FIELD_DIM_Y*nz*0.5));
	//~ fprintf(pipe, "set cbrange[-300:500]\n");
	fprintf(pipe, "set grid\n");
	fprintf(pipe, "set title 't=%.2f'\n",dt+DT);
	fprintf(pipe, "splot 'fdata.dat' u (($1-%f)*%f):(($2-%f)*%f):3 matrix with image noti\n",N*BLOCK_DIM_X*THREAD_DIM_X*0.5,FIELD_DIM_X/(BLOCK_DIM_X*THREAD_DIM_X),nz*BLOCK_DIM_Y*THREAD_DIM_Y*0.5,FIELD_DIM_Y/(BLOCK_DIM_Y*THREAD_DIM_Y));
	fflush(pipe);
	usleep(50000);
    return(0);

}





int main(){
	
	// timer variables	
	struct timeval t0,t1;
	
	// load waveforms
	float4 *wf, *wf_d;
	wf = (float4 *)malloc( NPULSES*RECLEN*NELS*sizeof(float4) );
	loadWF(wf); 
	hipMalloc( &wf_d, RECLEN*NELS*sizeof(float4) );
	hipMemcpy( wf_d, &wf[RECLEN*NELS], RECLEN*NELS*sizeof(float4), hipMemcpyHostToDevice ); // copy to gpu
	
	// load array coords to __constant__ memory on GPU
	loadArray();
	
	// allocate memory for signal intensity field calcs, initialize to 0, and put it on the GPU
	float *sig_field_host, *sig_field_pinned, *sig_field_d;
	int nmemb = (BLOCK_DIM_X*THREAD_DIM_X)*(BLOCK_DIM_Y*THREAD_DIM_Y)*(BLOCK_DIM_Z*THREAD_DIM_Z);
	unsigned int sig_field_size = nmemb*sizeof(float);
	sig_field_host = (float *)malloc(sig_field_size);
	hipHostMalloc((void **)&sig_field_pinned,sig_field_size, hipHostMallocDefault);
	hipMalloc((void **)&sig_field_d,sig_field_size);	
	memset(sig_field_host,0,sig_field_size); // initialize sig_field to 0
	memcpy(sig_field_pinned,sig_field_host,sig_field_size); // copy to pinned memory
	hipMemcpy(sig_field_d,sig_field_host,sig_field_size,hipMemcpyHostToDevice); // copy to gpu

	// allocate memory for the spatial coordinates of the calculation grid, populate it, and put it on the GPU
	float4 *xyz_h, *xyz_d;
	int nmembxyz = (BLOCK_DIM_X*THREAD_DIM_X)*(BLOCK_DIM_Y*THREAD_DIM_Y)*(BLOCK_DIM_Z*THREAD_DIM_Z);
	unsigned int xyz_size = nmembxyz*sizeof(float4);
	xyz_h = (float4 *)malloc(xyz_size);
	hipMalloc((void **)&xyz_d,xyz_size);
	setFieldLocsXYZ(xyz_h);
	hipMemcpy(xyz_d,xyz_h,xyz_size,hipMemcpyHostToDevice); // copy to gpu
	
	// setup cuda blocks/threads
	dim3 num_blocks(BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z);
	dim3 num_threads(THREAD_DIM_X, THREAD_DIM_Y, THREAD_DIM_Z);
	
	FILE *pipe = popen("gnuplot -persist","w");
	// calculate the signal intenisty in the field
	gettimeofday(&t0,NULL);
	for(float dt=-25.0;dt<5.1;dt+=0.2){
		resetSignalIntensityField <<<num_blocks, num_threads>>> ( sig_field_d );	
		getSignalIntensityField <<<num_blocks, num_threads>>> ( sig_field_d , wf_d, xyz_d, dt);
		hipMemcpy(sig_field_host,sig_field_d,sig_field_size,hipMemcpyDeviceToHost);	
		plotData(sig_field_host, pipe, dt);
	}
	gettimeofday(&t1,NULL);
	printf("calc time = %d us\n",t1.tv_usec-t0.tv_usec);
	
	sleep(2);
	fclose(pipe);
	// write the results to file
	FILE *fname;
	fname = fopen("sig_field_bin","wb");	
	fwrite(sig_field_host,sig_field_size,nmemb,fname);
	fclose(fname);

	// free allocated memory
	hipFree(wf_d);  free(wf);	
	hipFree(sig_field_d); 
	hipHostFree(sig_field_pinned);
	free(sig_field_host);
	hipFree(xyz_d); free(xyz_h);
	
	return 0;
}





